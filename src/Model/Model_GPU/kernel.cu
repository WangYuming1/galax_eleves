
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	 unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	 for(int j=0; j<n_particles;j++){
		if (i==j)
			continue;
		else{
			float3 distVector={0.0f, 0.0f,0.0f};
			distVector.x=positionsGPU[j].x-positionsGPU[i].x;
			distVector.y=positionsGPU[j].y-positionsGPU[i].y;
			distVector.z=positionsGPU[j].z-positionsGPU[i].z;
			float dijSqr=distVector.x*distVector.x+distVector.y*distVector.y+distVector.z*distVector.z;
			float dij=min(10.0f/(dijSqr * std::sqrt(dijSqr)),10.0f); 
			accelerationsGPU[i].x+=distVector.x*dij*massesGPU[j];
			accelerationsGPU[i].y+=distVector.y*dij*massesGPU[j];
			accelerationsGPU[i].z+=distVector.z*dij*massesGPU[j];
		}
	 }
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	 unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	 velocitiesGPU[i].x	+=	accelerationsGPU[i].x*2.0f;
	 velocitiesGPU[i].y	+=	accelerationsGPU[i].y*2.0f;
	 velocitiesGPU[i].z	+=	accelerationsGPU[i].z*2.0f;
	 positionsGPU[i].x	+=	velocitiesGPU[i].x*0.1f;
	 positionsGPU[i].y	+=	velocitiesGPU[i].y*0.1f;
	 positionsGPU[i].z	+=	velocitiesGPU[i].z*0.1f;
	
}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU